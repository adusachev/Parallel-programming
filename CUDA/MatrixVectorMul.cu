#include "hip/hip_runtime.h"
#include <iostream>

#include "matrix_functions.h"
#include "WriteResults.h"



__global__
void MatrixVectorMul(float* A, float* x, float* y, int width) {
	/* 
		Matrix-vector multiplication A * x = y
	*/
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    y[tid] = .0f;

    for (int k = 0; k < width; k++) {
        y[tid] += A[tid * width + k] * x[k];
    }
}


int main(int argc, char *argv[]) {

	int height = 1024; int width = 3840;
    // int height = 32; int width = 16;

	float *h_A = new float[height * width];  // matrix
    float *h_x = new float[width];  // vector
    float *h_y = new float[height];  // result vector

    // fill array
    for (int i = 0; i < width; i++) {
        h_x[i] = i; 
    }

	// EyeMatrix(h_A, height, width);
	RandomMatrix(h_A, height, width, 10);
	// save_matrix(h_A, height, width, "A.txt");


	float* d_A;
	float* d_x;
	float* d_y;
	hipMalloc(&d_A, sizeof(float) * height * width);
    hipMalloc(&d_x, sizeof(float) * width);
    hipMalloc(&d_y, sizeof(float) * height);

    hipMemcpy(d_A, h_A, sizeof(float) * height * width, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, sizeof(float) * width, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, sizeof(float) * height, hipMemcpyHostToDevice);

	int blockSize = atoi(argv[1]);  // get blocksize from command line
	// int blockSize = 32;

	int num_blocks = (height + blockSize - 1) / blockSize;
	std::cout << "blockSize = " << blockSize << "; num_blocks = " << num_blocks << std::endl;

	// measure calculations time
	hipEvent_t start, end;
	float milliseconds;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);


    MatrixVectorMul<<<num_blocks, blockSize>>>(d_A, d_x, d_y, width);

	hipEventRecord(end);  // end time measure

    hipMemcpy(h_y, d_y, sizeof(float) * height, hipMemcpyDeviceToHost);

	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);


	std::cout << "Time elapsed: " << milliseconds << " ms " << std::endl;


	hipFree(d_A);
	hipFree(d_x);
	hipFree(d_y);

	delete[] h_A;
	delete[] h_x;
	delete[] h_y;

	write_results(blockSize, milliseconds);

	return 0;
}
