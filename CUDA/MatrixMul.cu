
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>


void write_results(int block_size, double time, std::string filename="./results.csv") {
    /*
    	Write block_size and time values to the end of the file "filename"
    */
    std::ofstream out; 
    out.open(filename, std::ios::app);
    out << block_size << ", " << time << "\n";
    out.close();
}



void EyeMatrix(float* matrix, int height, int width) {
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			if (i == j) {
				matrix[i * width + j] = 1;
			} else {
				matrix[i * width + j] = 0;
			}
		}
	}
}

void OnesMatrix(float* matrix, int height, int width) {
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			matrix[i * width + j] = 1;
		}
	}
}


void save_matrix(float* matrix, int height, int width, std::string filename="./generated_matrix.csv") {
	std::ofstream out; 
	out.open(filename, std::ios::app);
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			out << matrix[i * width + j] << " ";
		}
	}
	out << "\n";
}


void RandomMatrix(float* matrix, int height, int width, int max_num) {
	/*
		Fill matrix with random ints from 0 to (max_num-1)
	*/
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			matrix[i * width + j] = rand() % max_num;
		}
	}
}

void PrintMatrix(float *matrix, int height, int width) {

	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			std::cout << i << " " << j << " " << matrix[i * width + j] << "\n";
		}
	}
}


__global__
void MatrixMul(float* A, float* B, float* C, int mid_size) {
	/* 
		Matrix multiplication A * B = C
	*/
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // line num
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // column num

    int height = blockDim.x * gridDim.x;
    int width = blockDim.y * gridDim.y;

    C[i * width + j] = .0f;

    for (int k = 0; k < mid_size; ++k) {
        C[i * width + j] += A[i * mid_size + k] * B[k * width + j];
    }
}


int main(int argc, char *argv[]) {
	float *h_A;
	float *h_B;
	float *h_C;
	int A_height = 128; int A_width = 384;
	int B_height = 384; int B_width = 256;
	int C_height = 128; int C_width = 256;
	int mid_size = 384;

	// int A_height = 5; int A_width = 6;
	// int B_height = 6; int B_width = 7;
	// int C_height = 5; int C_width = 7;
	// int mid_size = 6;

	h_A = new float[A_height * A_width];
	h_B = new float[B_height * B_width];
	h_C = new float[C_height * C_width];

	// EyeMatrix(h_A, A_height, A_width);
	// EyeMatrix(h_B, B_height, B_width);
	RandomMatrix(h_A, A_height, A_width, 12);
	RandomMatrix(h_B, B_height, B_width, 12);

	save_matrix(h_A, A_height, A_width, "A.txt");
	save_matrix(h_B, B_height, B_width, "B.txt");

    // PrintMatrix(h_A, A_height, A_width);

	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc(&d_A, sizeof(float) * A_height * A_width);
	hipMalloc(&d_B, sizeof(float) * B_height * B_width);
	hipMalloc(&d_C, sizeof(float) * C_height * C_width);

    hipMemcpy(d_A, h_A, sizeof(float) * A_height * A_width, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * B_height * B_width, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeof(float) * C_height * C_width, hipMemcpyHostToDevice);

	// 2D blocks and grid
	int BS = atoi(argv[1]);  // get blocksize from command line
	// int BS = 16;
	int blockSize_x = BS;
	int blockSize_y = BS;

	// want:  block_dim.x * num_blocks.x = height_C - size od matrix C
    //        block_dim.y * num_blocks.y = width_C
	int numBlocks_x = (C_height + blockSize_x - 1) / blockSize_x;
	int numBlocks_y = (C_width + blockSize_y - 1) / blockSize_y;

	std::cout << "numBlocks_x = " << numBlocks_x << "; numBlocks_y = " << numBlocks_y << std::endl;
	std::cout << "block_dim.x * num_blocks.x = " << blockSize_x * numBlocks_x 
			  << " <= " << C_height << " = height of C matrix" << std::endl;
	std::cout << "block_dim.y * num_blocks.y = " << blockSize_y * numBlocks_y 
			  << " <= " << C_width << " = width of C matrix" << std::endl;

	dim3 block_size(blockSize_x, blockSize_y);
	dim3 num_blocks(numBlocks_x, numBlocks_y);

	// measure calculations time
	hipEvent_t start, end;
	float milliseconds;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);


    MatrixMul<<<num_blocks, block_size>>>(d_A, d_B, d_C, mid_size);

	hipEventRecord(end);  // end time measure

    hipMemcpy(h_C, d_C, sizeof(float) * C_height * C_width, hipMemcpyDeviceToHost);

	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
	std::cout << "Time elapsed: " << milliseconds << " ms " << std::endl;

	save_matrix(h_C, C_height, C_width, "C.txt");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	write_results(blockSize_x, milliseconds);

	return 0;
}
