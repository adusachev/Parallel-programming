#include "hip/hip_runtime.h"
#include <iostream>

#define BLOCK_SIZE 256


#include "matrix_functions.h"
#include "WriteResults.h"







__global__ void KernelMatrixAdd(int height, int width, float* A, float* B, float* result) {
    /* 
		Matrix sum: A + B = C
	*/
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // line num
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // column num 
    
    result[i * width + j] = A[i * width + j] + B[i * width + j];
    
}



int main(int argc, char *argv[]) {
	float *h_A;
	float *h_B;
	float *h_C;

    int height = 1280;
    int width = 2560;
    // int height = 3;
    // int width = 3;

	h_A = new float[height * width];  // выделяем матрицы как flatten массивы
	h_B = new float[height * width];
	h_C = new float[height * width];

	EyeMatrix(h_A, height, width);
	EyeMatrix(h_B, height, width);
	// OnesMatrix(h_A, height, width);
	// OnesMatrix(h_B, height, width);

    // PrintMatrix(h_A, height, width);

	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc(&d_A, sizeof(float) * height * width);
	hipMalloc(&d_B, sizeof(float) * height * width);
	hipMalloc(&d_C, sizeof(float) * height * width);

    hipMemcpy(d_A, h_A, sizeof(float) * height * width, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * height * width, hipMemcpyHostToDevice);

    // measure calculations time
	hipEvent_t start, end;
	float milliseconds;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

    // 2D blocks and grid
	int BS = atoi(argv[1]);  // get blocksize from command line
	// int BS = 16;
	int blockSize_x = BS;
	int blockSize_y = BS;
    dim3 block_size(blockSize_x, blockSize_y);  // each block has 16x16 threads

	// want:  block_dim.x * num_blocks.x = height
    //        block_dim.y * num_blocks.y = width
	int numBlocks_x = (height + blockSize_y - 1) / blockSize_x;
	int numBlocks_y = (width + blockSize_x - 1) / blockSize_x;
	dim3 num_blocks(numBlocks_x, numBlocks_y);

	// dim3 num_blocks(8, 16);  // 8x16 blocks

    std::cout << "numBlocks_x = " << numBlocks_x << "; numBlocks_y = " << numBlocks_y << std::endl;
	std::cout << "block_dim.x * num_blocks.x = " << blockSize_x * numBlocks_x << " <= " << height << " = height" << std::endl;
	std::cout << "block_dim.y * num_blocks.y = " << blockSize_y * numBlocks_y << " <= " << width << " = width" << std::endl;


    KernelMatrixAdd<<<num_blocks, block_size>>>(height, width, d_A, d_B, d_C);

    hipEventRecord(end);
    hipMemcpy(h_C, d_C, sizeof(float) * height * width, hipMemcpyDeviceToHost);

    hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "Time elapsed: " << milliseconds << " ms " << std::endl;

    PrintMatrix(h_C, height, width);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	write_results(blockSize_x, milliseconds);

	return 0;
}
