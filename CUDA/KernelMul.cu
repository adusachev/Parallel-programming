#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "WriteResults.h"




__global__
void KernelMul(int n, float* x, float* y, float* res) {
	/*  
        Поэлементное произведение двух массивов x и y
	*/
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = tid; i < n; i += stride) {
		res[i] = x[i] * y[i];
	}
}


int main(int argc, char *argv[]) {
	int n = 1 << 26;  // 2**28

	// step 1: allocate Host memory
	float *h_x = new float[n];
	float *h_y = new float[n];
	float *h_res = new float[n];

	// step 2: allocate Device memory
	float *d_x;
	float *d_y;
	float *d_res;
	int nbytes = n * sizeof(float);  // size in bytes
	hipMalloc(&d_x, nbytes);
	hipMalloc(&d_y, nbytes);
	hipMalloc(&d_res, nbytes);

	// fill Host arrays
	for (int i = 0; i < n; i++) {
		h_x[i] = 2.0f;
		h_y[i] = 3.0f;
	}

	// step 3: copy arrays from Host to Device (size in bytes!)
	hipMemcpy(d_x, h_x, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, nbytes, hipMemcpyHostToDevice);

	// measure calculations time
	hipEvent_t start, end;
	float milliseconds;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);


	// step 4: run calculations
	int blockSize = atoi(argv[1]);  // get blocksize from command line
	// int blockSize = 256;
	int numBlocks = (n + blockSize - 1) / blockSize;

	KernelMul<<<numBlocks, blockSize>>>(n, d_x, d_y, d_res);


	// end time measure
	hipEventRecord(end);

	// step 5: copy calc result from Device to Host
	hipMemcpy(h_res, d_res, nbytes, hipMemcpyDeviceToHost);


	hipEventSynchronize(end);  // (!)

	hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "Time elapsed: " << milliseconds << " ms " << std::endl;

	// step 6: free Host memory
	delete[] h_x;
	delete[] h_y;
	delete[] h_res;

	// step 7: free Device memory
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_res);

	write_results(blockSize, milliseconds);


	return 0;
}
