
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 256


void EyeMatrix(float* matrix, int height, int width) {
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			if (i == j) {
				matrix[i * width + j] = 1;
			} else {
				matrix[i * width + j] = 0;
			}
		}
	}
}

void OnesMatrix(float* matrix, int height, int width) {
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			matrix[i * width + j] = 1;
		}
	}
}


void PrintMatrix(float *matrix, int height, int width) {

	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			std::cout << "(i, j)=(" << i << ", " << j << ") --> " << matrix[i * width + j] << "\n";
		}
	}
}




__global__ void KernelMatrixAdd(int height, int width, float* A, float* B, float* result) {
    /* 
		Matrix sum: A + B = C
	*/
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // координаты потоков по осям х и у
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // так как каждый элемент обрабатывается своим потоком, цикл не нужен
    // for (int k = 0; k < height * width; k++) {
    result[i * width + j] = A[i * width + j] + B[i * width + j];
    // }
}



int main() {
	float *h_A;
	float *h_B;
	float *h_C;

    int height = 128;
    int width = 256;
    // int height = 3;
    // int width = 3;

	h_A = new float[height * width];  // выделяем матрицы как flatten массивы
	h_B = new float[height * width];
	h_C = new float[height * width];

	EyeMatrix(h_A, height, width);
	EyeMatrix(h_B, height, width);

    // PrintMatrix(h_A, height, width);


	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc(&d_A, sizeof(float) * height * width);
	hipMalloc(&d_B, sizeof(float) * height * width);
	hipMalloc(&d_C, sizeof(float) * height * width);
    hipMemcpy(d_A, h_A, sizeof(float) * height * width, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * height * width, hipMemcpyHostToDevice);


    // measure calculations time
	hipEvent_t start, end;
	float milliseconds;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);


    // двумерные блоки и потоки
    dim3 num_blocks(8, 16);  // 8x16 блоков 
    dim3 block_size(16, 16);  // в каждом блоке 16х16 потоков

    // хотим: block_dim.x * num_blocks.x = width
    //        block_dim.y * num_blocks.y = height
    // 8*16=128, 16*16=256 - как чсило строк и столбцов в матрице С

    KernelMatrixAdd<<<num_blocks, block_size>>>(height, width, d_A, d_B, d_C);

    hipEventRecord(end);
    hipMemcpy(h_C, d_C, sizeof(float) * height * width, hipMemcpyDeviceToHost);

    hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "Time elapsed: " << milliseconds << " ms " << std::endl;

    PrintMatrix(h_C, height, width);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	return 0;
}
